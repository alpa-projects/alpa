#include"kernel.h"
#include<stdio.h>

namespace kernel{

void offloadToHost(hipStream_t stream, void **buffers, const char *opaque, size_t opaque_len) {
    const float *x = reinterpret_cast<const float *>(buffers[0]);
    float *key = reinterpret_cast<float *>(buffers[1]);
    const swapHelper *s = reinterpret_cast<const swapHelper*>(opaque);
    auto allocInfo = s->manager->allocMemory(s->TensorSize);

    hipMemcpy((float *)allocInfo.second,
        x,
        s->TensorSize,
        hipMemcpyDeviceToHost
    );
    hipMemcpy((float *)key,
        &allocInfo.first,
        sizeof(float),
        hipMemcpyHostToDevice);
}

void uploadToDevice(hipStream_t stream, void **buffers, const char *opaque, size_t opaque_len) {
    const float *key = reinterpret_cast<const float*>(buffers[0]);
    float *result = reinterpret_cast<float *>(buffers[1]);
    const swapHelper *s = reinterpret_cast<const swapHelper *>(opaque);
    float host_key;
    hipMemcpy(&host_key, key, sizeof(float), hipMemcpyDeviceToHost);

    auto entry = s->manager->getMemInfo(host_key);

    hipMemcpy(result,
        entry.first,
        entry.second,
        hipMemcpyHostToDevice
    );
}

};  //end namespace kernel
