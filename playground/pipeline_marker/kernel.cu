#include"kernel.h"
#include<stdio.h>

namespace kernel{
void pipelineMarker(hipStream_t stream, void **buffers, const char *opaque, size_t opaque_len) {
    const int64_t *sizes = reinterpret_cast<const int64_t *> opaque;
    for (size_t i = 0; i < opaque_len; i++) {
        printf("%lld %lld\n", i, sizes[i])
    }
    const float *x = reinterpret_cast<const float *>(buffers[0]);
    float *result = reinterpret_cast<float *>(buffers[1]);
    hipMemset(result, 0, 1);
}
};  //end namespace kernel
