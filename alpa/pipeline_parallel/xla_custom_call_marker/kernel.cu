#include "kernel.h"
#include <stdio.h>

namespace kernel {

void identity(hipStream_t stream, void **buffers, const char *opaque, size_t opaque_len) {
    const int64_t *sizes = reinterpret_cast<const int64_t *>(opaque);
    size_t n_inputs = opaque_len / sizeof(int64_t);
    for (size_t i = 0; i < n_inputs; i++) {
        const void *input = reinterpret_cast<const void *>(buffers[i]);
        void *output = reinterpret_cast<void *>(buffers[i + n_inputs]);
        if (input != output) {
            printf("WARNING: The inputs and outputs of idenity marker are not aliases\n");
            hipMemcpy(output, input, sizes[i], hipMemcpyDeviceToDevice);
        }
    }
}

};  // end namespace kernel
